#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <time.h>

#define M 5
#define N 3

// cuComplex or cuDoubleComplex
#define CPLX hipDoubleComplex

void init_rand(int *data,int size){
    for (int i = 0; i < size; ++i)
        data[i] = rand() %100;
}

void print_cplx(int *data,int m,int n){
  for(int i=0;i<m;i++){
    for(int j=0;j<n;j++){
      printf("%d ",data[i*n+ j]);
    }
    printf("\n");
  }
    printf("\n");
}

__global__ void transpose(int*X,int*X_T,int m,int n);


int main(void) 
{
    int *d_a, *d_b, *d_c; 
    int *h_a, *h_b, *h_c; 

    time_t t;
    srand(time(&t));

    int memsize = sizeof(int)*M*N;
 
    h_a = (int*)malloc(memsize);
    h_b = (int*)malloc(memsize);
    h_c = (int*)malloc(memsize);

    init_rand(h_a,M*N);

    print_cplx(h_a,M,N);
 

    hipMalloc((void **)&d_a, memsize);
    hipMalloc((void **)&d_b, memsize);
    hipMalloc((void **)&d_c, memsize);


    hipMemcpy(d_a, h_a, memsize, hipMemcpyHostToDevice);

    transpose<<<M,N>>>(d_a,d_b,M,N);

    hipMemcpy(h_b, d_b, memsize, hipMemcpyDeviceToHost);

    print_cplx(h_b,N,M);
/************************************************************/
    hipMemcpy(d_a, h_b, memsize, hipMemcpyHostToDevice);

    //Roll Back
    transpose<<<N,M>>>(d_a,d_b,N,M);

    hipMemcpy(h_a, d_b, memsize, hipMemcpyDeviceToHost);

    print_cplx(h_a,M,N);

    free(h_a);free(h_b);free(h_c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}


__global__ void transpose(int*X,int*X_T,int m,int n){

   int idxX = threadIdx.x ;
   int idxY = blockIdx.x ;

   int idx_in  = idxX + n*idxY;
   int idx_out = idxY + m*idxX;
   X_T[idx_out] = X[idx_in];
}

