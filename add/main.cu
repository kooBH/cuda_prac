#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) 
{
    *c = *a + *b;
}


int main(void) 
{
    int a, b, c; 
    int *d_a, *d_b, *d_c; 
 
    hipMalloc((void **)&d_a, sizeof(int));
    hipMalloc((void **)&d_b, sizeof(int));
    hipMalloc((void **)&d_c, sizeof(int));

    a = 3;
    b = 4;

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    add<<<1,1>>>(d_a, d_b, d_c);

    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    printf("%d + %d = %d\n",a,b,c);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}


