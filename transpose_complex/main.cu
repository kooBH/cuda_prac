#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <time.h>

#define M 5
#define N 3

// cuComplex or cuDoubleComplex
#define CPLX hipDoubleComplex

void init_crand(int *data,int size){
    for (int i = 0; i < size*2; ++i)
        data[i] = rand() %100 - 50;
}

void print_cplx(int *data,int m,int n){
  for(int i=0;i<m;i++){
    for(int j=0;j<n;j++){
      printf("%3d%+3di ",data[i*n*2+ j+j],data[i*n*2+j+j+1]);
    }
    printf("\n");
  }
    printf("\n");
}

__global__ void ctranspose(int*X,int*X_T,int m,int n);


int main(void) 
{
    int *d_a, *d_b, *d_c; 
    int *h_a, *h_b, *h_c; 

    time_t t;
    srand(time(&t));

    int memsize = sizeof(int)*M*N*2;
 
    h_a = (int*)malloc(memsize);
    h_b = (int*)malloc(memsize);
    h_c = (int*)malloc(memsize);

    init_crand(h_a,M*N);

    print_cplx(h_a,M,N);
 

    hipMalloc((void **)&d_a, memsize);
    hipMalloc((void **)&d_b, memsize);
    hipMalloc((void **)&d_c, memsize);


    hipMemcpy(d_a, h_a, memsize, hipMemcpyHostToDevice);

    ctranspose<<<M,N>>>(d_a,d_b,M,N);

    hipMemcpy(h_b, d_b, memsize, hipMemcpyDeviceToHost);

    print_cplx(h_b,N,M);
/************************************************************/
    hipMemcpy(d_a, h_b, memsize, hipMemcpyHostToDevice);

    //Roll Back
    ctranspose<<<N,M>>>(d_a,d_b,N,M);

    hipMemcpy(h_a, d_b, memsize, hipMemcpyDeviceToHost);

    print_cplx(h_a,M,N);

    free(h_a);free(h_b);free(h_c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}


__global__ void ctranspose(int*X,int*X_T,int m,int n){

   int idxX = threadIdx.x*2 ;
   int idxY = blockIdx.x*2 ;

   int idx_in  = idxX + n*idxY;
   int idx_out = idxY + m*idxX;
   X_T[idx_out] = X[idx_in];
   X_T[idx_out+1] = X[idx_in+1];
}

