#include "hip/hip_runtime.h"

int *d_a,*d_b,*d_c;
int *h_a,*h_b,*h_c;

__host__ void init(){
  h_a = (int*)malloc(sizeof(int));
  h_b = (int*)malloc(sizeof(int));
  h_c = (int*)malloc(sizeof(int));

  *h_c=0;

  hipMalloc((void**)&d_a,sizeof(int));
  hipMalloc((void**)&d_b,sizeof(int));
  hipMalloc((void**)&d_c,sizeof(int));

  hipMemcpy(d_c,h_c,sizeof(int),hipMemcpyHostToDevice);
}

__host__  void assign(int a,int b){
  *h_a = a;
  *h_b = b;

  hipMemcpy(d_a,h_a,sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(d_b,h_b,sizeof(int),hipMemcpyHostToDevice);
}

__global__ void add(int *a,int *b,int *c){
  *c += *a + *b;
}

__host__ void process(){
  add<<<1,1>>>(d_a,d_b,d_c);
}

__host__ int get(){
  hipMemcpy(h_c,d_c,sizeof(int),hipMemcpyDeviceToHost);
  return *h_c;
}



