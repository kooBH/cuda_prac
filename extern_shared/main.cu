#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"

#define M 8
#define N 12

extern __shared__ int shared[];

__global__ void func(int*data, int m,int n){
  int * t_s = shared;
//  __shared__ int t_s[12];

  if(threadIdx.x==0){
     memcpy(t_s + blockIdx.x*n,data + blockIdx.x*n,sizeof(int)*n);
  }
  __syncthreads();

  data[blockIdx.x*n + threadIdx.x] = 
  t_s[blockIdx.x*n + threadIdx.x+1];
}


int main(){

int h_d[M*N];
int *d_d;

  for(int i=0;i<M*N;i++)
    h_d[i]=i+1;
 
  for(int i=0;i<M;i++){
    for(int j=0;j<N;j++)
      printf("%2d ",h_d[i*N+j]);
    printf("\n");
  }
  printf("\n");

  hipMalloc((void**)&d_d,sizeof(int)*M*N);
  hipMemcpy(d_d,h_d,sizeof(int)*M*N,hipMemcpyHostToDevice);


  func<<<M,N-1,sizeof(int)*M*N>>>(d_d,M,N);

  hipDeviceSynchronize();

  memset(h_d,0,sizeof(int)*M*N);
  hipMemcpy(h_d,d_d,sizeof(int)*M*N,hipMemcpyDeviceToHost);

  for(int i=0;i<M;i++){
    for(int j=0;j<N;j++)
      printf("%2d ",h_d[i*N+j]);
    printf("\n");
  }
  printf("\n");



  return 0;

}
