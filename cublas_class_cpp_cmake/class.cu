#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <stdio.h>

__global__ void some_func(double*data,double value);

class A{
  double *d_a;
  double *h_a;
  const int size  = 10;
  int memsize;

  public : 
  A();
  void Print();
};


__global__ void some_func(double*data,double value){

      data[threadIdx.x] = value;
}


A::A(){

  memsize = size * sizeof(double);


  h_a = (double*)malloc(memsize);
  hipMalloc((void**)&d_a,memsize);

  some_func<<<1,size>>>(d_a,1);

  hipMemcpy(&h_a,d_a,memsize,hipMemcpyDeviceToHost);

}

void A::Print(){
   for(int i=0;i<size;i++)
     printf("%lf ",h_a[i]);
   printf("\n");
}


