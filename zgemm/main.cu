#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <stdio.h>
#include <iostream>
#include <time.h>

// hipComplex or hipDoubleComplex
#define CPLX hipDoubleComplex

void init_rand(double*data,int size){
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (double)RAND_MAX;
}

void print_cplx(double*data,int size){
  for(int i=0;i<size;i++){
    for(int j=0;j<size;j++){
      printf("%lf %lf | ",data[i*(2*size)+ 2*j],data[i*(2*size)+ 2*j+1]);
    }
    printf("\n");
  }
    printf("\n");
}


int main(void) 
{
    const int size = 4;
    double*d_a, *d_b, *d_c; 
    double*h_a, *h_b, *h_c; 

    time_t t;
    srand(time(&t));

    int memsize = sizeof(double)*size*size*2;
 
    h_a = (double*)malloc(memsize);
    h_b = (double*)malloc(memsize);
    h_c = (double*)malloc(memsize);

    init_rand(h_a,size*size*2);
    init_rand(h_b,size*size*2);

    print_cplx(h_a,size);
    print_cplx(h_b,size);


    hipMalloc((void **)&d_a, memsize);
    hipMalloc((void **)&d_b, memsize);
    hipMalloc((void **)&d_c, memsize);


    hipMemcpy(d_a, h_a, memsize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, memsize, hipMemcpyHostToDevice);


   hipblasHandle_t handle;
   hipblasCreate(&handle);

    hipDoubleComplex alpha {1.0,0.0};
    hipDoubleComplex beta {0.0,0.0};

    std::cout<<"alpha : "<<alpha.x<<" "<<alpha.y<<"\n";

    hipblasZgemm(handle,
        HIPBLAS_OP_N,HIPBLAS_OP_N,
        size,size,size,
        &alpha,
        (CPLX*)d_a, size,
        (CPLX*)d_b, size,
        &beta,
        (CPLX*)d_c, size);

    hipMemcpy(h_c, d_c, memsize, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    print_cplx(h_c,size);

    free(h_a);free(h_b);free(h_c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}


